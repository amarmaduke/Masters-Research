#include "hip/hip_runtime.h"
#include "euler.h"
#include "force.h"
#include <stdio.h>
#include <vector>
#include <iostream>

#define cudaH2D hipMemcpyHostToDevice
#define cudaD2H hipMemcpyDeviceToHost
#define cudaD2D hipMemcpyDeviceToDevice

#ifdef _ERROR_
#define checkError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n",
		hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
#else
	#define checkError(ans) ans
#endif

void snapshot(vector<triple> save, double *x, double *y, double *s)
{
	double *first = new double[size];
	double *second = new double[size];
	double *third = new double[2];

	checkError(hipMemcpy(first,x,sizeof(double)*size,cudaD2H));
	checkError(hipMemcpy(second,y,sizeof(double)*size,cudaD2H));
	checkError(hipMemcpy(third,s,sizeof(double)*2,cudaD2H));

	triple temp_trip(first,second,third);
	save.push_back(temp_trip);
}

__global__
void vector_sum(double * const out_x,
								double * const out_y,
								double * const out_s,
								const double * const a_x,
								const double * const a_y,
								const double * const a_s,
								const double * const b_x,
								const double * const b_y,
								const double * const b_s)
{
	int index = threadIdx.x;
	out_x[index] = a_x[index] + b_x[index];
	out_y[index] = a_y[index] + b_y[index];
	out_s[index] = a_s[index] + b_s[index];
}

__global__
void update(double * const update_x,
						double * const update_y,
						double * const update_s,
						double * const force_x,
						double * const force_y,
						double * const force_s,
						double h)
{
	int index = threadIdx.x;
	double x = update_x[index];
	double y = update_y[index];
	double s = update_s[index];

	update_x[index] = x + h*force_x[index];
	update_y[index] = y + h*force_y[index];
	update_s[index] = s + h*force_s[index];
}

std::vector<triple>
eulers_method(const double * const x,
							const double * const y,
							const double * const s,
							const double * const delta,
							double t_start,
							double t_end,
							double h,
							int save,
							parameter p)
{
	std::vector<triple> store;

	double *force_x, *force_y, *force_s;
	double *update_x, *update_y, *update_s;
	int size = p.n * p.m;

	checkError(hipMalloc(&force_x,sizeof(double)*size));
	checkError(hipMalloc(&force_y,sizeof(double)*size));
	checkError(hipMalloc(&force_s,sizeof(double)*2));
	checkError(hipMalloc(&update_x,sizeof(double)*size));
	checkError(hipMalloc(&update_y,sizeof(double)*size));
	checkError(hipMalloc(&update_s,sizeof(double)*2));

	checkError(hipMemcpy(update_x,x,sizeof(double)*size,cudaD2D));
	checkError(hipMemcpy(update_y,y,sizeof(double)*size,cudaD2D));
	checkError(hipMemcpy(update_s,s,sizeof(double)*2,cudaD2D));

	double t = t_start;

	int total_points = (int)((t_end - t_start)/h + 1);
	int sampling_rate = total_points / save;

	if ( sampling_rate <= 0 )
	{
		sampling_rate = 1;
	}
	int count = 0;

	hipStream_t s1, s2;
	checkError(hipStreamCreate(&s1));
	checkError(hipStreamCreate(&s2));

	while(t <= t_end)
	{
		if(count % sampling_rate == 0)
		{
			snapshot(store,update_x,update_y,update_s);

			printf("current time: %f, final time: %f",t,t_end);
			std::cout << std::endl;
		}

		force(force_x,force_y,force_s,update_x,update_y,update_s,delta,p);
		update<<<1,size>>>(update_x,update_y,update_s,force_x,force_y,force_s,h);

		++count;
		t+=h;
	}

	snapshot(store,update_x,update_y,update_s);

	checkError(hipStreamDestroy(s1));
	checkError(hipStreamDestroy(s2));

	checkError(hipFree(force_x));
	checkError(hipFree(force_y));
	checkError(hipFree(force_s));
	checkError(hipFree(update_x));
	checkError(hipFree(update_y));
	checkError(hipFree(update_s));

	return store;
}


std::vector<triple>
euler_heun_adaptive(const double * const x,
										const double * const y,
										const double * const s,
										const double * const delta,
										double t_start,
										double t_end,
										double h,
										int save,
										double tolerance,
										parameter p)
{
	std::vector<triple> store;

	double *temp_x, *temp_y, *temp_s;
	double *update_x, *update_y, *update_s;
	int size = p.n * p.m;

	checkError(hipMalloc(&temp_x,sizeof(double)*size));
	checkError(hipMalloc(&temp_y,sizeof(double)*size));
	checkError(hipMalloc(&temp_s,sizeof(double)*2));
	checkError(hipMalloc(&update_x,sizeof(double)*size));
	checkError(hipMalloc(&update_y,sizeof(double)*size));
	checkError(hipMalloc(&update_s,sizeof(double)*2));

	checkError(hipMemcpy(update_x,x,sizeof(double)*size,cudaD2D));
	checkError(hipMemcpy(update_y,y,sizeof(double)*size,cudaD2D));
	checkError(hipMemcpy(update_s,s,sizeof(double)*2,cudaD2D));

	double *k1_x, *k1_y, *k1_s, *k2_x, *k2_y, *k2_s;
	checkError(hipMalloc(&k1_x,sizeof(double)*size));
	checkError(hipMalloc(&k1_y,sizeof(double)*size));
	checkError(hipMalloc(&k1_s,sizeof(double)*size));
	checkError(hipMalloc(&k2_x,sizeof(double)*size));
	checkError(hipMalloc(&k2_y,sizeof(double)*size));
	checkError(hipMalloc(&k2_s,sizeof(double)*size));

	double t = t_start;

	int total_points = (int)((t_end - t_start)/h + 1);
	int sampling_rate = total_points / save;

	if ( sampling_rate <= 0 )
	{
		sampling_rate = 1;
	}
	int count = 0;

	double error = 100;
	while(t <= t_end)
	{
		if(count % sampling_rate == 0)
		{
			snapshot(store,update_x,update_y,update_s);

			printf("current time: %f, final time: %f",t,t_end);
			std::cout << std::endl;
		}

		while(error > tolerance)
		{
			force(k1_x,k1_y,k1_s,update_x,update_y,update_s,delta,p);
			update<<<1,size>>>(temp_x,temp_y,temp_s,k1_x,k1_y,k1_s,h);
			force(k2_s,k2_y,k2_s,temp_x,temp_y,temp_s,delta,p);
			vector_sum<<<1,size>>>(temp_x,temp_y,temp_s,k1_x,k1_y,k1_s,k2_x,k2_y,k2_s);
			update<<<1,size>>>(update_x,update_y,update_s,temp_x,temp_y,temp_s,h/2);
		}

		++count;
		t+=h;
	}

	snapshot(store,update_x,update_y,update_s);

	checkError(hipFree(temp_x));
	checkError(hipFree(temp_y));
	checkError(hipFree(temp_s));
	checkError(hipFree(update_x));
	checkError(hipFree(update_y));
	checkError(hipFree(update_s));
	checkError(hipFree(k1_x));
	checkError(hipFree(k1_y));
	checkError(hipFree(k1_s));
	checkError(hipFree(k2_x));
	checkError(hipFree(k2_y));
	checkError(hipFree(k2_s));

	return store;
}
