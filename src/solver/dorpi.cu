#include "dorpi.h"
#include "../simulation/force.h"

double y_constants[7][7] = {
{       (1/2),            0,           0,         0,              0,          0,    0},
{      (3/40),       (9/40),           0,         0,              0,          0,    0},
{     (44/45),     (-56/15),      (32/9),         0,              0,          0,    0},
{(19372/6561),(-25360/2187),(64448/6561),(-212/729),              0,          0,    0},
{ (9017/3168),    (-355/33),(46732/5247),  (49/176),  (-5103/18656),          0,    0},
{    (35/384),            0,  (500/1113), (125/192),   (-2187/6784),    (11/84),    0},
{(5179/57600),            0,(7571/16695), (393/640),(-92097/339200),(187/2100),(1/40)}};

double t_constants[6] = {(1/5), (3/10), (4/5), (8/9), 1, 1};

#define cudaH2D hipMemcpyHostToDevice
#define cudaD2H hipMemcpyDeviceToHost
#define cudaD2D hipMemcpyDeviceToDevice

void SNAPSHOT(std::vector<triple>& save, double *x, double *y, double *s, int size)
{
  double *first = new double[size];
  double *second = new double[size];
  double *third = new double[2];

  check_error(hipMemcpy(first,x,sizeof(double)*size,cudaD2H));
  check_error(hipMemcpy(second,y,sizeof(double)*size,cudaD2H));
  check_error(hipMemcpy(third,s,sizeof(double)*2,cudaD2H));

  triple temp_trip(first,second,third);
  save.push_back(temp_trip);
}

template<typename T>
void print_v(int n, T * x)
{
	std::cout << "(";
	for(int i = 0; i < n; ++i)
	{
		std::cout << x[i];
		if(i + 1 != n)
			std::cout << " ";
	}
	std::cout << ")";
}

double * linc_s(hipblasHandle_t handle, int count, int size,
              std::vector<double>& constants, std::vector<double *>& vectors)
{
  // Preconditions
  assert(constants.size() == vectors.size());
  count = count < 1 || count > vectors.size() ? vectors.size() : count;

  std::vector<int> iter(count);
  std::vector<double *> vectors_copy(vectors.size());
  std::vector<hipStream_t> streams(count);

  double *buffer;
  check_error(hipMalloc(&buffer, sizeof(double)*size*count));
  for(int i = 0; i < count; ++i)
  {
    // Setup iterator
    iter.push_back(i);

    // Setup Streams
    hipStream_t stream;
    check_error(hipStreamCreate(&stream));
    streams.push_back(stream);

    // Grab vector
    double *temp = buffer + i*size;
    hipblasSetStream(handle,stream);
    hipblasDcopy(handle, size, vectors[i], 1, temp, 1);
    vectors_copy.push_back(temp);
  }
  hipDeviceSynchronize();

  // Compute linear compination
  int N = count, m, j = 0;
  while(N > 1)
  {
    m = N % 2 == 0 ? N : N - 1;
    for(int i = 0; i < m; i+=2, ++j)
    {
      double alpha = constants[iter[i+1]]/constants[iter[i]];
      hipblasSetStream(handle,streams[i]);
      hipblasDaxpy(handle, count, &alpha, vectors_copy[iter[i+1]], 1,
                  vectors_copy[iter[i]], 1);
      iter[j] = iter[i];
    }
    hipDeviceSynchronize();
    if(N % 2 != 0)
      iter[j] = iter[m];
    N = (N+1)/2;
    j = 0;
  }

  double a = constants[iter[0]];
  hipblasDscal(handle, count, &a, vectors[0], 1);

  // Grab output and clean
  double * out = vectors[0];
  check_error(hipFree(buffer));
  for(int i = 0; i < count; ++i)
  {
    hipStreamDestroy(streams[i]);
  }

  return out;
}
