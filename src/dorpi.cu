#include "hip/hip_runtime.h"
#include "dorpi.h"
#include "force.h"
#include <stdio.h>
#include <stdarg.h>
#include <stdlib.h>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define cudaH2D hipMemcpyHostToDevice
#define cudaD2H hipMemcpyDeviceToHost
#define cudaD2D hipMemcpyDeviceToDevice

#ifdef _ERROR_
#define checkError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr,"GPUassert: %s %s %d\n",
    hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
#else
  #define checkError(ans) ans
#endif

void SNAPSHOT(std::vector<triple>& save, double *x, double *y, double *s, int size)
{
  double *first = new double[size];
  double *second = new double[size];
  double *third = new double[2];

  checkError(hipMemcpy(first,x,sizeof(double)*size,cudaD2H));
  checkError(hipMemcpy(second,y,sizeof(double)*size,cudaD2H));
  checkError(hipMemcpy(third,s,sizeof(double)*2,cudaD2H));

  triple temp_trip(first,second,third);
  save.push_back(temp_trip);
}

template<typename T>
void print_v(int n, T * x)
{
	std::cout << "(";
	for(int i = 0; i < n; ++i)
	{
		std::cout << x[i];
		if(i + 1 != n)
			std::cout << " ";
	}
	std::cout << ")";
}

// count = # of vectors passed to linc
// size = # of elements in each vector
// ... is assumed to have count (constant,vector) pairs
double * linc(hipblasHandle_t handle, int count, int size, ...)
{
  va_list ap;
	double ** vectors, * constants;
  int * iter;
	
	vectors = new double*[count];
	constants = new double[count];
	iter = new int[count];

  // Parse variadic input and store copies
  va_start(ap,size);
	for(int i = 0; i < count; ++i)
  {
    // Setup iterator
    iter[i] = i;

    // Grab constant
		double c_temp = va_arg(ap, double);
		constants[i] = c_temp;

    // Grab vector
    checkError(hipMalloc(&vectors[i], sizeof(double)*size));
		double * v_temp = va_arg(ap, double *);
		
		hipblasDcopy(handle, size, v_temp, 1, vectors[i], 1);
	}
  va_end(ap);

  // Compute linear compination
  int N = count, m, j = 0;
  while(N > 1)
  {
		std::cout << "iter: " << std::endl;
		print_v(count,iter);
		std::cout << std::endl;
    m = N % 2 == 0 ? N : N - 1;
    for(int i = 0; i < m; i+=2, ++j)
    {
      double a = constants[iter[i+1]]/constants[iter[i]];
      
			double *x, *y;
			x = new double[size]; y = new double[size];
			hipMemcpy(x,vectors[iter[i+1]],sizeof(double)*size,cudaD2H);
			hipMemcpy(y,vectors[iter[i]],sizeof(double)*size,cudaD2H);
			
			std::cout << " Inner linc Loop: " << std::endl;
			std::cout << a;
			print_v(size,x);
			std::cout << " + ";
			print_v(size,y);
			std::cout << std::endl;

			hipblasDaxpy(handle, count, &a, vectors[iter[i+1]], 1, vectors[iter[i]], 1);
      iter[j] = iter[i];
    }
    if(N % 2 != 0)
      iter[j] = iter[m];
    N = (N+1)/2;
    j = 0;
		std::cout << " Outer liner loop: " << std::endl;
		std::cout << "N: " << N << std::endl;
  }
	
	double a = constants[iter[0]];
	hipblasDscal(handle, count, &a, vectors[0], 1);

  // Grab output and clean
  double * out = vectors[0];
  for(int i = 1; i < count; ++i)
  {
    hipFree(vectors[i]);
  }
  delete constants;
  delete iter;
	delete vectors;

  return out;
}

void test()
{
	int N = 3;
	double *h_x, *h_y, *h_z, *h_a, *h_b, *h_r;
	double *d_x, *d_y, *d_z, *d_a, *d_b, *d_r;

	h_x = new double[N];
	h_y = new double[N];
	h_z = new double[N];
	h_a = new double[N];
	h_b = new double[N];
	h_r = new double[N];

	for(int i = 0; i < N; ++i)
	{
		h_x[i] = 1;
		h_y[i] = 1;
		h_z[i] = 1;
		h_a[i] = 1;
		h_b[i] = 1;
	}

	hipMalloc(&d_x, sizeof(double)*N);
	hipMalloc(&d_y, sizeof(double)*N);
	hipMalloc(&d_z, sizeof(double)*N);
	hipMalloc(&d_a, sizeof(double)*N);
	hipMalloc(&d_b, sizeof(double)*N);
	
	hipMemcpy(d_x,h_x,sizeof(double)*N,cudaH2D);
	hipMemcpy(d_y,h_y,sizeof(double)*N,cudaH2D);
	hipMemcpy(d_z,h_z,sizeof(double)*N,cudaH2D);
	hipMemcpy(d_a,h_a,sizeof(double)*N,cudaH2D);
	hipMemcpy(d_b,h_b,sizeof(double)*N,cudaH2D);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	std::cout << "1";
	print_v(N,h_x);
	std::cout << std::endl;
	std::cout << "2";
	print_v(N,h_y);
	std::cout << std::endl;
	std::cout << "3";
	print_v(N,h_z);
	std::cout << std::endl;
	std::cout << "2";
	print_v(N,h_a);
	std::cout << std::endl;
	std::cout << "3";
	print_v(N,h_b);
	std::cout << std::endl;


	d_r = linc(handle,5,N,1.0,d_x,2.0,d_y,3.0,d_z,2.0,d_a,3.0,d_b);

	hipMemcpy(h_r,d_r,sizeof(double)*N,cudaD2H);
	
	for(int i = 0; i < N; ++i)
	{
		std::cout << h_r[i] << " ";
	}
	std::cout << std::endl;
}

__global__
void construct_next_step( double * const out_x,
                          double * const out_y,
                          double * const out_s,
                          const double * const x,
                          const double * const y,
                          const double * const s,
                          const double * const k1_x,
                          const double * const k1_y,
                          const double * const k1_s,
                          const double * const k2_x,
                          const double * const k2_y,
                          const double * const k2_s,
                          const double * const k3_x,
                          const double * const k3_y,
                          const double * const k3_s,
                          const double * const k4_x,
                          const double * const k4_y,
                          const double * const k4_s,
                          const double * const k5_x,
                          const double * const k5_y,
                          const double * const k5_s,
                          const double * const k6_x,
                          const double * const k6_y,
                          const double * const k6_s,
                          const double * const a,
                          const double h)
{
  int index = threadIdx.x;
  out_x[index] = x[index] + h*(a[0]*k1_x[index] + a[1]*k2_x[index]
                 + a[2]*k3_x[index] + a[3]*k4_x[index]
                 + a[4]*k5_x[index] + a[5]*k6_x[index]);
  out_y[index] = y[index] + h*(a[0]*k1_y[index] + a[1]*k2_y[index]
                 + a[2]*k3_y[index] + a[3]*k4_y[index]
                 + a[4]*k5_y[index] + a[5]*k6_y[index]);
  out_s[0] = s[0] + h*(a[0]*k1_s[0] + a[1]*k2_s[0]
                 + a[2]*k3_s[0] + a[3]*k4_s[0]
                 + a[4]*k5_s[0] + a[5]*k6_s[0]);
  out_s[1] = s[1] + h*(a[0]*k1_s[1] + a[1]*k2_s[1]
                 + a[2]*k3_s[1] + a[3]*k4_s[1]
                 + a[4]*k5_s[1] + a[5]*k6_s[1]);
}

__global__
void construct_error_step(double * const out_x,
                          double * const out_y,
                          double * const out_s,
                          const double * const k1_x,
                          const double * const k1_y,
                          const double * const k1_s,
                          const double * const k2_x,
                          const double * const k2_y,
                          const double * const k2_s,
                          const double * const k3_x,
                          const double * const k3_y,
                          const double * const k3_s,
                          const double * const k4_x,
                          const double * const k4_y,
                          const double * const k4_s,
                          const double * const k5_x,
                          const double * const k5_y,
                          const double * const k5_s,
                          const double * const k6_x,
                          const double * const k6_y,
                          const double * const k6_s,
                          const double * const k7_x,
                          const double * const k7_y,
                          const double * const k7_s,
                          const double * const b)
{
  int index = threadIdx.x;
  out_x[index] = b[0]*k1_x[index] + b[1]*k2_x[index] + b[2]*k3_x[index]
                 + b[3]*k4_x[index] + b[4]*k5_x[index] + b[5]*k6_x[index]
                 + b[6]*k7_x[index];
  out_y[index] = b[0]*k1_y[index] + b[1]*k2_y[index] + b[2]*k3_y[index]
                 + b[3]*k4_y[index] + b[4]*k5_y[index] + b[5]*k6_y[index]
                 + b[6]*k7_y[index];
  out_s[0] = b[0]*k1_s[0] + b[1]*k2_s[0] + b[2]*k3_s[0] + b[3]*k4_s[0]
                 + b[4]*k5_s[0] + b[5]*k6_s[0] + b[6]*k7_s[0];
  out_s[1] = b[0]*k1_s[1] + b[1]*k2_s[1] + b[2]*k3_s[1] + b[3]*k4_s[1]
                 + b[4]*k5_s[1] + b[5]*k6_s[1] + b[6]*k7_s[1];
}

std::vector<triple>
dormand_prince( const double * const x,
                const double * const y,
                const double * const s,
                const double * const delta,
                double t_start,
                double t_end,
                double h,
                int save,
                double tolerance,
                parameter p)
{
  std::vector<triple> store;

  double *temp_x, *temp_y, *temp_s;
  double *update_x, *update_y, *update_s;
  int size = p.n * p.m;

  checkError(hipMalloc(&temp_x,sizeof(double)*size));
  checkError(hipMalloc(&temp_y,sizeof(double)*size));
  checkError(hipMalloc(&temp_s,sizeof(double)*2));
  checkError(hipMalloc(&update_x,sizeof(double)*size));
  checkError(hipMalloc(&update_y,sizeof(double)*size));
  checkError(hipMalloc(&update_s,sizeof(double)*2));

  checkError(hipMemcpy(update_x,x,sizeof(double)*size,cudaD2D));
  checkError(hipMemcpy(update_y,y,sizeof(double)*size,cudaD2D));
  checkError(hipMemcpy(update_s,s,sizeof(double)*2,cudaD2D));

  double *zero, *zer2;
  checkError(hipMalloc(&zero,sizeof(double)*size));
  checkError(hipMemset(zero,0,sizeof(double)*size));
  checkError(hipMalloc(&zer2,sizeof(double)*2));
  checkError(hipMemset(zer2,0,sizeof(double)*2));

  double *k1_x, *k1_y, *k1_s, *k2_x, *k2_y, *k2_s, *k3_x, *k3_y, *k3_s;
  double *k4_x, *k4_y, *k4_s, *k5_x, *k5_y, *k5_s, *k6_x, *k6_y, *k6_s;
  double *k7_x, *k7_y, *k7_s, *err_x, *err_y, *err_s;
  checkError(hipMalloc(&k1_x,sizeof(double)*size));
  checkError(hipMalloc(&k1_y,sizeof(double)*size));
  checkError(hipMalloc(&k1_s,sizeof(double)*2));
  checkError(hipMalloc(&k2_x,sizeof(double)*size));
  checkError(hipMalloc(&k2_y,sizeof(double)*size));
  checkError(hipMalloc(&k2_s,sizeof(double)*2));
  checkError(hipMalloc(&k3_x,sizeof(double)*size));
  checkError(hipMalloc(&k3_y,sizeof(double)*size));
  checkError(hipMalloc(&k3_s,sizeof(double)*2));
  checkError(hipMalloc(&k4_x,sizeof(double)*size));
  checkError(hipMalloc(&k4_y,sizeof(double)*size));
  checkError(hipMalloc(&k4_s,sizeof(double)*2));
  checkError(hipMalloc(&k5_x,sizeof(double)*size));
  checkError(hipMalloc(&k5_y,sizeof(double)*size));
  checkError(hipMalloc(&k5_s,sizeof(double)*2));
  checkError(hipMalloc(&k6_x,sizeof(double)*size));
  checkError(hipMalloc(&k6_y,sizeof(double)*size));
  checkError(hipMalloc(&k6_s,sizeof(double)*2));
  checkError(hipMalloc(&k7_x,sizeof(double)*size));
  checkError(hipMalloc(&k7_y,sizeof(double)*size));
  checkError(hipMalloc(&k7_s,sizeof(double)*2));
  checkError(hipMalloc(&err_x,sizeof(double)*size));
  checkError(hipMalloc(&err_y,sizeof(double)*size));
  checkError(hipMalloc(&err_s,sizeof(double)*2));

  double *a1, *a2, *a3, *a4, *a5, *a6, *a7;
  checkError(hipMalloc(&a1,sizeof(double)*7));
  checkError(hipMalloc(&a2,sizeof(double)*7));
  checkError(hipMalloc(&a3,sizeof(double)*7));
  checkError(hipMalloc(&a4,sizeof(double)*7));
  checkError(hipMalloc(&a5,sizeof(double)*7));
  checkError(hipMalloc(&a6,sizeof(double)*7));
  checkError(hipMalloc(&a7,sizeof(double)*7));

  // Butcher Tableau constants

  a1[0] = 1/5; a1[1] = a1[2] = a1[3] = a1[4] = a1[5] = a1[6] = 0;
  a2[0] = 3/40; a2[1] = 9/40; a2[2] = a2[3] = a2[4] = a2[5] = a2[6] = 0;
  a3[0] = 44/45; a3[1] = -56/15; a3[2] = 32/9; a3[3] = a3[4] = a3[5] = a3[6] =0;
  a4[0] = 19372/6561; a4[1] = -25360/2187; a4[2] = 64448/6561;
    a4[3] = -212/729; a4[4] = a4[5] = a4[6] = 0;
  a5[0] = 9017/3168; a5[1] = -355/33; a5[2] = 46732/5247; a5[3] = 49/176;
    a5[4] = -5103/18656; a5[5] = a5[6] = 0;
  a6[0] = 35/384; a6[1] = 0; a6[2] = 500/1113; a6[3] = 125/192;
    a6[4] = -2187/6784; a6[5] = 11/84; a6[6] = 0;
  a7[0] = 5179/57600; a7[1] = 0; a7[2] = 7571/16695; a7[3] = 393/640;
    a7[4] = -92097/339200; a7[5] = 187/2100; a7[6] = 1/40;

  double t = t_start;

  int total_points = (int)((t_end - t_start)/h + 1);
  int sampling_rate = total_points / save;

  if ( sampling_rate <= 0 )
  {
    sampling_rate = 1;
  }
  int count = 0;

  hipblasHandle_t handle;

  double error = 100;
  while(t <= t_end)
  {
    if(count % sampling_rate == 0)
    {
      SNAPSHOT(store,update_x,update_y,update_s,size);

      printf("current time: %f, final time: %f",t,t_end);
      std::cout << std::endl;
    }

    while(error > tolerance)
    {
      // k1 Step
      force(k1_x,k1_y,k1_s,update_x,update_y,update_s,delta,p);
      construct_next_step<<<1,size>>>
          (temp_x,temp_y,temp_s,update_x,update_y,update_s,k1_x,k1_y,k1_s,
          zero,zero,zer2,zero,zero,zer2,zero,zero,zer2,zero,zero,zer2,
          zero,zero,zer2,a1,h);

      // k2 Step
      force(k2_x,k2_y,k2_s,temp_x,temp_y,temp_s,delta,p);
      construct_next_step<<<1,size>>>
          (temp_x,temp_y,temp_s,update_x,update_y,update_s,k1_x,k1_y,k1_s,
          k2_x,k2_y,k2_s,zero,zero,zer2,zero,zero,zer2,zero,zero,zer2,
          zero,zero,zer2,a2,h);

      // k3 Step
      force(k3_x,k3_y,k3_s,temp_x,temp_y,temp_s,delta,p);
      construct_next_step<<<1,size>>>
          (temp_x,temp_y,temp_s,update_x,update_y,update_s,k1_x,k1_y,k1_s,
          k2_x,k2_y,k2_s,k3_x,k3_y,k3_s,zero,zero,zer2,zero,zero,zer2,
          zero,zero,zer2,a3,h);

      // k4 Step
      force(k4_x,k4_y,k4_s,temp_x,temp_y,temp_s,delta,p);
      construct_next_step<<<1,size>>>
          (temp_x,temp_y,temp_s,update_x,update_y,update_s,k1_x,k1_y,k1_s,
          k2_x,k2_y,k2_s,k3_x,k3_y,k3_s,k4_x,k4_y,k4_s,zero,zero,zer2,
          zero,zero,zer2,a4,h);

      // k5 Step
      force(k5_x,k5_y,k5_s,temp_x,temp_y,temp_s,delta,p);
      construct_next_step<<<1,size>>>
          (temp_x,temp_y,temp_s,update_x,update_y,update_s,k1_x,k1_y,k1_s,
          k2_x,k2_y,k2_s,k3_x,k3_y,k3_s,k4_x,k4_y,k4_s,k5_x,k5_y,k5_s,
          zero,zero,zer2,a5,h);

      // k6 Step
      force(k6_x,k6_y,k6_s,temp_x,temp_y,temp_s,delta,p);
      construct_next_step<<<1,size>>>
          (temp_x,temp_y,temp_s,update_x,update_y,update_s,k1_x,k1_y,k1_s,
          k2_x,k2_y,k2_s,k3_x,k3_y,k3_s,k4_x,k4_y,k4_s,k5_x,k5_y,k5_s,
          k6_x,k6_y,k6_s,a6,h);

      // k7 Step
      force(k7_x,k7_y,k7_s,temp_x,temp_y,temp_s,delta,p);

      construct_error_step<<<1,size>>>
          (err_x,err_y,err_s,k1_x,k1_y,k1_s,k2_x,k2_y,k2_s,k3_x,k3_y,k3_s,
          k4_x,k4_y,k4_s,k5_x,k5_y,k5_s,k6_x,k6_y,k6_s,k7_x,k7_y,k7_s,a6);
      /*
      checkError(hipMemcpy(heun,update_x,sizeof(double)*size,cudaD2D));
      checkError(hipMemcpy(heun+size,update_y,sizeof(double)*size,cudaD2D));
      checkError(hipMemcpy(heun+size*2,update_s,sizeof(double)*2,cudaD2D));

      // Error Handling
      vector_subtraction<<<1,size>>>(vec,euler,heun);
      hipblasCreate(&handle);
      hipblasSnrm2(handle,2*size+2,vec,1,&error); // Euclidean Norm

      if(error > tolerance)
        h /= 2; */
    }

    ++count;
    t+=h;
  }

  SNAPSHOT(store,update_x,update_y,update_s,size);

  // Wall of Destruction

  checkError(hipFree(temp_x)); checkError(hipFree(update_x));
  checkError(hipFree(temp_y)); checkError(hipFree(update_y));
  checkError(hipFree(temp_s)); checkError(hipFree(update_s));
  checkError(hipFree(k1_x)); checkError(hipFree(k5_x));
  checkError(hipFree(k1_y)); checkError(hipFree(k5_x));
  checkError(hipFree(k1_s)); checkError(hipFree(k5_x));
  checkError(hipFree(k3_x)); checkError(hipFree(k6_x));
  checkError(hipFree(k3_y)); checkError(hipFree(k6_x));
  checkError(hipFree(k3_s)); checkError(hipFree(k6_x));
  checkError(hipFree(k2_x)); checkError(hipFree(k7_x));
  checkError(hipFree(k2_y)); checkError(hipFree(k7_x));
  checkError(hipFree(k2_s)); checkError(hipFree(k7_x));
  checkError(hipFree(k4_x));
  checkError(hipFree(k4_y));
  checkError(hipFree(k4_s));
  checkError(hipFree(a1)); checkError(hipFree(a4)); checkError(hipFree(a6));
  checkError(hipFree(a2)); checkError(hipFree(a5)); checkError(hipFree(a6));
  checkError(hipFree(a3)); checkError(hipFree(zero));
  checkError(hipFree(zer2)); checkError(hipFree(err_x));
  checkError(hipFree(err_y)); checkError(hipFree(err_s));

  return store;
}
