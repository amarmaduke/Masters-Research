#include "hip/hip_runtime.h"
#include "force.h"
#include <stdio.h>


// Concept Binary Operator Op
//		requires (T a, T b){ op(a,b) -> T }
template<typename Op>
__global__
void scan(value_type* v, size_t size, Op op)
{
  int index = blockDim.x * blockIdx.x + threadIdx.x;
	value_type val = v[index];
	for(int i = 1; i < size; ++i)
	{
		if(index + i < size)
		{
			value_type temp = v[index+i];
			v[index+i] = op(temp,val);
		}
		__syncthreads();
	}
}

template<typename Op>
__global__
void combine(value_type* out, value_type* a, value_type* b, size_t size, Op op)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if(index < size)
	{
		value_type temp_xa = a[index];
		value_type temp_ya = a[index+size];
		value_type temp_xb = b[index];
		value_type temp_yb = b[index+size];
		out[index] = op(temp_xa,temp_xb);
		out[index+size] = op(temp_ya,temp_yb);
	}
}

__device__
void position(int& j, int& i, const int ptr, const parameter& p)
{
  j = ptr / p.n;
  i = (ptr % p.n);
}

__global__
void compute_other( value_type* const out_x,
                    value_type* const out_y,
                    value_type* const out_sx,
                    value_type* const out_sy,
                    const value_type* const in_x,
                    const value_type* const in_y,
                    const value_type* const in_s,
                    const parameter p)
{
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int size = p.n*p.m;

  //if(index >= size)
  //  return;

  int j, i;
  int n = p.n;
  value_type beta = p.beta;
  value_type len = p.len;
  value_type gamma = p.gamma;
  value_type epsilon = p.epsilon;
  value_type sigma = p.sigma;
	value_type pressure = p.pressure;
  value_type* delta = p.delta;

  position(j,i,index,p);

  value_type xp = i == 0 ? delta[j] : in_x[index-1];
  value_type xpp = i == 0 || i == 1 ? delta[j] : in_x[index-2];
  value_type yp = i == 0 ? 0 : in_y[index-1];
  value_type ypp =  i == 0 || i == 1 ?
              ( i == 0 ? -len : 0) : in_y[index-2];

  value_type xn = (index % n) + 1 < n ? in_x[index+1] : NAN;
  value_type xnn = (index % n) + 2 < n ? in_x[index+2] : NAN;
  value_type yn = (index % n) + 1 < n ? in_y[index+1] : NAN;
  value_type ynn = (index % n) + 2 < n ? in_y[index+2] : NAN;

  value_type x = in_x[index];
  value_type y = in_y[index];

  value_type lp = sqrt((xp-xpp)*(xp-xpp) + (yp-ypp)*(yp-ypp));
  value_type l = sqrt((x-xp)*(x-xp) + (y-yp)*(y-yp));
  value_type ln = sqrt((xn-x)*(xn-x) + (yn-y)*(yn-y));
  value_type lnn = sqrt((xnn-xn)*(xnn-xn) + (ynn-yn)*(ynn-yn));

  // Bending Force

  value_type xd_f = (xn - x)*(xnn - xn);
  value_type yd_f = (yn - y)*(ynn - yn);
  value_type xd_c = (x - xp)*(xn - x);
  value_type yd_c = (y - yp)*(yn - y);
  value_type xd_b = (xp - xpp)*(x - xp);
  value_type yd_b = (yp - ypp)*(y - yp);

  value_type product_f = xd_f + yd_f;
  value_type product_c = xd_c + yd_c;
  value_type product_b = xd_b + yd_b;

  value_type b_3_t1 = 4.0*(lnn/ln)*product_f;
	value_type b_3_t2 = 4.0*(lnn*ln);
	value_type b_3_b = (lnn*ln + product_f)*(lnn*ln+product_f);
	value_type b_3x = (b_3_t1*(x-xn) - b_3_t2*(xn-xnn))/b_3_b;
	value_type b_3y = (b_3_t1*(y-yn) - b_3_t2*(yn-ynn))/b_3_b;

	value_type b_2_t1x = 4.0*(l/ln*(x-xn) + ln/l*(x-xp))*product_c;
	value_type b_2_t2x = 4.0*ln*l*(xp-2.0*x+xn);
	value_type b_2_t1y = 4.0*(l/ln*(y-yn) + ln/l*(y-yp))*product_c;
	value_type b_2_t2y = 4.0*ln*l*(yp-2.0*y+yn);
	value_type b_2_b = (ln*l + product_c)*(ln*l + product_c);
	value_type b_2x = (b_2_t1x - b_2_t2x)/b_2_b;
	value_type b_2y = (b_2_t1y - b_2_t2y)/b_2_b;

	value_type b_1_t1 = 4.0*(lp/l)*product_b;
	value_type b_1_t2 = 4.0*l*lp;
	value_type b_1_b = (l*lp + product_b)*(l*lp + product_b);
	value_type b_1x = (b_1_t1*(x-xp) - b_1_t2*(xp-xpp))/b_1_b;
	value_type b_1y = (b_1_t1*(y-yp) - b_1_t2*(yp-ypp))/b_1_b;

	b_1x = isnan(b_1x) ? 0 : b_1x;
	b_2x = isnan(b_2x) ? 0 : b_2x;
	b_3x = isnan(b_3x) ? 0 : b_3x;

	b_1y = isnan(b_1y) ? 0 : b_1y;
	b_2y = isnan(b_2y) ? 0 : b_2y;
	b_3y = isnan(b_3y) ? 0 : b_3y;

  value_type bending_x = beta*(b_1x + b_2x + b_3x);
  value_type bending_y = beta*(b_1y + b_2y + b_3y);

  // Extensible Spring Force

  value_type e_forward = (ln - len)/ln;
  value_type e_backward = (l - len)/l;

  value_type e_forward_x = e_forward*2.0*(x-xn);
  value_type e_backward_x = e_backward*2.0*(x-xp);
  value_type e_forward_y = e_forward*2.0*(y-yn);
  value_type e_backward_y = e_backward*2.0*(y-yp);

  e_forward_x = isnan(e_forward_x) ? 0 : e_forward_x;
  e_backward_x = isnan(e_backward_x) ? 0 : e_backward_x;
  e_forward_y = isnan(e_forward_y) ? 0 : e_forward_y;
  e_backward_y = isnan(e_backward_y) ? 0 : e_backward_y;

  value_type extensible_x = gamma*(e_forward_x + e_backward_x);
  value_type extensible_y = gamma*(e_forward_y + e_backward_y);

  // Lower substrate vdW pressure

  value_type p1, p2, p4, p5, p7, p8, p11, p13;
  p1 = sigma / y;
  p2 = p1*p1;
  p4 = p2*p2;
  p5 = p4*p1;
  p11 = p5*p5*p1;

  value_type vdW_y = -(pressure*PI*epsilon)*(2.0*p11-4.0*p5);

  // Upper substrate vdW

  value_type s_x = in_s[0];
  value_type s_y = in_s[1];
  value_type s_vdW_sx = 0, s_vdW_sy = 0, s_vdW_x = 0, s_vdW_y = 0;
  int sub_count = p.sub_count;
  value_type sub_h = p.sub_h;

  for(int k = 0; k < sub_count; ++k)
  {
    value_type x_ = s_x + k*sub_h;
    value_type y_ = s_y;

    value_type xps = x_ - x;
    value_type yps = y_ - y;
    value_type dist = sqrt(xps*xps + yps*yps);

    value_type temp_x = xps/dist;
    value_type temp_y = yps/dist;

    p1 = sigma / dist;
    p2 = p1*p1;
    p4 = p2*p2;
    p7 = p4*p2*p1;
    p8 = p7*p1;
    p13 = p8*p4*p1;
    value_type LJval = -(12.0*epsilon/sigma)*(p13-p7);

    s_vdW_x = s_vdW_x + LJval*temp_x;
    s_vdW_y = s_vdW_y + LJval*temp_y;

    s_vdW_sx = s_vdW_sx - LJval*temp_x;
    s_vdW_sy = s_vdW_sy - LJval*temp_y;
  }

  // Lower substrate vdW

  value_type os_x = p.osub;
  value_type os_vdW_x = 0, os_vdW_y = 0;
  int osub_count = p.osub_count;
  value_type osub_h = p.osub_h;

  for(int k = 0; k < osub_count; ++k)
  {
    value_type x_ = os_x + k*osub_h;
    value_type y_ = 0;

    value_type xps = x_ - x;
    value_type yps = y_ - y;
    value_type dist = sqrt(xps*xps + yps*yps);

    value_type temp_x = xps/dist;
    value_type temp_y = yps/dist;

    p1 = sigma / dist;
    p2 = p1*p1;
    p4 = p2*p2;
    p7 = p4*p2*p1;
    p8 = p7*p1;
    p13 = p8*p4*p1;
    value_type LJval = -(12.0*epsilon/sigma)*(p13-p7);

    os_vdW_x = os_vdW_x + LJval*temp_x;
    os_vdW_y = os_vdW_y + LJval*temp_y;
  }

  // Total Force

  value_type total_force_x = -(bending_x + extensible_x) + s_vdW_x + os_vdW_x;
  value_type total_force_y = -(bending_y + extensible_y + vdW_y) + s_vdW_y + os_vdW_y;

  total_force_x = -(bending_x + extensible_x) + s_vdW_x + os_vdW_x;
  total_force_y = -(bending_y + extensible_y + vdW_y) + s_vdW_y + os_vdW_y;

  //printf("index: %d, j: %d, i: %d\nxpp: %f, xp: %f, x: %f, xn: %f, xnn: %f\n ypp: %f, yp: %f, y: %f, yn: %f, ynn: %f\nb_x: %f, e_x: %f, vu_x: %f, vl_x: %f\nb_y: %f, e_y: %f, vp_y: %f, vu_y: %f, vl_y: %f\n",index,j,i,xpp,xp,x,xn,xnn,ypp,yp,y,yn,ynn,bending_x,extensible_x,s_vdW_x,os_vdW_x,bending_y,extensible_y,vdW_y,s_vdW_y,os_vdW_y);

  out_x[index] = total_force_x;
  out_y[index] = total_force_y;
  out_sx[index] = s_vdW_sx;
  out_sy[index] = s_vdW_sy;
}

__device__
value_type2 lennard_jones(value_type2 v, value_type2 v_,
                      int2 idx, int2 idx_, value_type2 acc,
                      value_type sigma, value_type epsilon)
{
  value_type xps = v.x - v_.x;
  value_type yps = v.y - v_.y;

  // Add machine epsilon to prevent 0 / 0 introducing a NaN
  // This is implemented strictly to avoid branching.
  value_type dist = sqrt(xps*xps + yps*yps);

  value_type temp_x = xps/dist;
  value_type temp_y = yps/dist;

  value_type p1 = sigma / dist;
  value_type p2 = p1*p1;
  value_type p4 = p2*p2;
  value_type p7 = p4*p2*p1;
  value_type p8 = p7*p1;
  value_type p13 = p8*p4*p1;
  value_type LJval = -(12.0*epsilon/sigma)*(p13-p7);

  // Condense j == j_ and (i == i_ or i == i_ + 1 or i == i_ - 1) via two
  // always positive continuous functions with zeros only at those points.
  //int s1 = ((idx.y - idx_.y)*(idx.y - idx_.y) - 1)*(idx.y - idx_.y)
  //         *((idx.y - idx_.y)*(idx.y - idx_.y) - 1)*(idx.y - idx_.y);
  //int s2 = (idx.x - idx_.x)*(idx.x - idx_.x);

  // Conditional execution instead of branching
  //int swtch = (s1 + s2 == 0);
  //int swtch = (abs(idx.y - idx.y) - 1)*(idx.y - idx_.y);
  bool swtch = idx.x == idx_.x
          and (idx.y == idx_.y or idx.y == idx_.y + 1 or idx.y == idx_.y - 1);

  acc.x += swtch? 0 : -LJval*temp_x;
  acc.y += swtch? 0 : -LJval*temp_y;

  return acc;
}

__device__
value_type2 tile_calculation( value_type2 v, int index, int index_,
                          value_type2 acc, parameter& p)
{
  int i;
  extern __shared__ value_type2 pos[];
  #pragma unroll 4
  for(i = 0; i < blockDim.x; ++i)
  {
    int2 idx, idx_;
    position(idx.x,idx.y,index,p);
    position(idx_.x,idx_.y,index_+i,p);
    acc = lennard_jones(v,pos[i],idx,idx_,acc,p.sigma,p.epsilon);
  }
  return acc;
}

__global__
void compute_n_body(value_type* const out,
                    const value_type* const in,
                    parameter p)
{
  extern __shared__ value_type2 pos[];
  int i, tile, index = blockIdx.x * blockDim.x + threadIdx.x;
  int size = p.n * p.m;
  value_type2 acc = {0.0, 0.0};
  value_type2 v = {in[index], in[index+size]};

  for(i = 0, tile = 0; i < size; i += K, ++tile)
  {
    int idx = tile * blockDim.x + threadIdx.x;
    value_type2 f = {in[idx], in[idx+size]};
    pos[threadIdx.x] = f;
    __syncthreads();
    acc = tile_calculation(v,index,tile*blockDim.x,acc,p);
    __syncthreads();
  }
  out[index] = acc.x;
  out[index+size] = acc.y;
}

void
force_functor::operator() ( const vector_type &x,
                            vector_type &dxdt,
                            const value_type dt)
{
  int size = this->state.n*this->state.m;
  int B = size%K != 0? size/K + 1 : size/K;
  dim3 block_other(B,1,1), thread_other(K,1,1);
  dim3 block_nbody(B,1,1), thread_nbody(K,1,1);

  const value_type* const in = x.data().get();
  value_type* const out = dxdt.data().get();

  hipStream_t s1, s2;
  hipEvent_t e1, e2;
  hipStreamCreate(&s1);
  hipStreamCreate(&s2);
  hipEventCreate(&e1);
  hipEventCreate(&e2);

  thrust::device_ptr<value_type> nbody, substrate;
  nbody = thrust::device_malloc<value_type>(2*size);
  substrate = thrust::device_malloc<value_type>(2*size);

  compute_other<<<block_other,thread_other,0,s1>>>
                ( out,out+size,
                  substrate.get(),substrate.get()+size,
                  in,in+size,in+2*size,
                  this->state);
  hipEventRecord(e1,s1);

  compute_n_body<<<block_nbody,thread_nbody,K*sizeof(value_type2),s2>>>
                  (nbody.get(),in,this->state);

  hipEventSynchronize(e1);

  value_type sub_x = thrust::reduce(substrate,substrate+size);
  value_type sub_y = thrust::reduce(substrate+size,substrate+2*size);

  hipDeviceSynchronize();

  thrust::transform(nbody,nbody+2*size,dxdt.data(),dxdt.data(),thrust::plus<value_type>());

  dxdt[2*size] = sub_x + this->state.mu;
  dxdt[2*size+1] = sub_y - this->state.lambda;

	/*
	for(int i = 0; i < 2*size+2; ++i)
	{
		std::cout << dxdt[i] << " ";
	} std::cout << std::endl;
	assert(false);
	*/

  hipStreamDestroy(s1);
  hipStreamDestroy(s2);
  hipEventDestroy(e1);
  thrust::device_free(nbody);
  thrust::device_free(substrate);
}

void
force_functor2::operator() ( const vector_type &x,
                            vector_type &dxdt,
                            const value_type dt)
{
  int size = this->state.n*this->state.m;
	int total_size = 2*size+2;
  int B = size%K != 0? size/K + 1 : size/K;
  dim3 block_other(B,1,1), thread_other(K,1,1);
  dim3 block_nbody(B,1,1), thread_nbody(K,1,1);

  const value_type* const in = x.data().get();
  value_type* const out = dxdt.data().get();

  hipStream_t s[SIM_COUNT];
  for(int i = 0; i < SIM_COUNT; ++i)
  {
    hipStreamCreate(&s[i]);
  }

  thrust::device_ptr<value_type> nbody, substrate;
  nbody = thrust::device_malloc<value_type>(2*size*SIM_COUNT);
  substrate = thrust::device_malloc<value_type>(2*size*SIM_COUNT);

	#pragma unroll
  for(int i = 0; i < SIM_COUNT; ++i)
  {
    compute_other<<<block_other,thread_other,0,s[i]>>>
                  ( out + i*total_size, out+size + i*total_size,
                    substrate.get() + i*2*size,
                    substrate.get()+size + i*2*size,
                    in + i*total_size, in+size + i*total_size,
                    in+2*size + i*total_size, this->state);

    //value_type sub_x = thrust::reduce(substrate+i*total_size,
    //                                  substrate+size+i*total_size);
    //value_type sub_y = thrust::reduce(substrate+size+i*total_size,
    //                                  substrate+2*size+i*total_size);

    compute_n_body<<<block_nbody,thread_nbody,K*sizeof(value_type2),s[i]>>>
                    (nbody.get() + i*2*size,
                      in + i * total_size,
                      this->state);

    hipStreamSynchronize(s[i]);

		value_type sub_x, sub_y;
		thrust::plus<value_type> op;
		scan<<<block_other,thread_other,0,s[i]>>>
					(substrate.get()+i*2*size, size, op);
		scan<<<block_other,thread_other,0,s[i]>>>
					(substrate.get()+size+i*2*size, size, op);
    
		combine<<<block_other,thread_other,0,s[i]>>>
					(out+i*total_size, nbody.get()+i*2*size, out+i*total_size, size, op);
		//thrust::transform(nbody+i*total_size,
    //                  nbody+2*size+i*total_size,
    //                  dxdt.data()+i*total_size,
    //                  dxdt.data()+i*total_size,
    //                  thrust::plus<value_type>());
    hipStreamSynchronize(s[i]);
	
		hipMemcpy(&sub_x,substrate.get()+size+i*2*size-1,sizeof(double),hipMemcpyDeviceToHost);
		hipMemcpy(&sub_y,substrate.get()+2*size+i*2*size-1,sizeof(double),hipMemcpyDeviceToHost);
	
		//sub_x = substrate[size+i*total_size];
		//sub_y = substrate[2*size+i*total_size];
    
		dxdt[2*size+i*total_size] = sub_x + this->mu[i];
    dxdt[2*size+1+i*total_size] = sub_y - this->lambda[i];
  }

  for(int i = 0; i < SIM_COUNT; ++i)
  {
    hipStreamDestroy(s[i]);
  }
  thrust::device_free(nbody);
  thrust::device_free(substrate);
}
